//Parallel Matrix-Matrix addition on GPU using CUDA
//Matrix dimensions are MxN
//C = A + B


#include <hip/hip_runtime.h>
#include<math.h>
#include<stdio.h>

#define M (8192)
#define N (4096)

//Parallel Matrix-Matrix addition kernel
__global__
void m_add(float *a, float *b, float *c) {
	int index = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < M && col < N) {
		index = row*N+col;
		c[index] = a[index] + b[index];
	}
	return;
}

int main(int argc, char **argv) {
	int index = 0;
	float *A, *B, *C;
	float max_err = 0.0f;
	int size = M * N * sizeof(float);

	//Define the Grid and Block dimensions
	dim3 threads_per_blk(16, 16);
	dim3 blocks_per_grid((N+threads_per_blk.x-1)/threads_per_blk.x,
											 (M+threads_per_blk.y-1)/threads_per_blk.y);

	//Allocate unified memory for linear 2D array
	hipMallocManaged(&A, size);
	hipMallocManaged(&B, size);
	hipMallocManaged(&C, size);

	//Initialize the MxN matrices
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			index = i*N+j;
			A[index] = 2.0f;
			B[index] = 3.0f;
			C[index] = 0.0f;
		}
	}

	//Launch the kernel on the GPU
	m_add<<<blocks_per_grid, threads_per_blk>>>(A, B, C);

	//Wait for GPU to finish
	hipDeviceSynchronize();

	//Verify GPU math
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			index = i*N+j;
			max_err = fmax(max_err, fabs(C[index]-(A[index] + B[index])));
		}
	}

	printf("Max error: %f\n", max_err);

	//Free memory
	hipFree(A);
	hipFree(B);
	hipFree(C);

	return 0;
}
