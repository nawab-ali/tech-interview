//Parallel SAXPY on GPU using CUDA
//z = ax + y


#include <hip/hip_runtime.h>
#include<math.h>
#include<stdio.h>

//CUDA SAXPY kernel
__global__
void saxpy(int n, float a, float *x, float *y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < n; i += stride) {
		y[i] = a * x[i] + y[i];
	}
	return;
}

int main() {
	float *x;
	float *y;
	int N = 1<<25;
	float a = 2.0f;
	float max_err = 0.0;
	int blocksize = 256;
	int nblocks = (N + blocksize - 1) / blocksize;

	//Allocate unified memory for vectors x, y
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	//Initialize x, y
	for (int i = 0; i < N; i++) {
		x[i] = 2.0f;
		y[i] = 3.0f;
	}

	//Run SAXPY kernel on the GPU
	saxpy<<<nblocks, blocksize>>>(N, a, x, y);

	//Wait for GPU to finish
	hipDeviceSynchronize();

	//Verify results
	for (int i = 0; i < N; i++) {
		max_err = fmax(max_err, fabs(y[i] - 7.0f));
	}

	printf("Max error: %f\n", max_err);

	//Free memory
	hipFree(x);
	hipFree(y);

	return 0;
}
