
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

//GPU kernel to add the elements of 2 arrays
__global__
void add(int n, float *x, float *y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < n; i += stride) {
		y[i] = x[i] + y[i];
	}
}

int main(void) {
	int N = 1<<20;
	float *x, *y;

	//Allocate unified memory
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	//Initialize x, y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	int blocksize = 256;
	int numblocks = (N + blocksize - 1)/blocksize;

	//Run kernel on 1M elements on the GPU
	add<<<numblocks, blocksize>>>(N, x, y);

	//Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	//Check for errors (all values should be 3.0f)
	float maxerror = 0.0f;
	for (int i = 0; i < N; i++) {
		maxerror = fmax(maxerror, fabs(y[i] - 3.0f));
	}
	std::cout << "Max error: " << maxerror << std::endl;

	//Free memory
	hipFree(x);
	hipFree(y);

  return 0;
}
