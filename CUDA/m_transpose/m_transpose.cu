//Matrix transpose on GPU using CUDA


#include <hip/hip_runtime.h>
#include<time.h>
#include<math.h>
#include<stdio.h>
#include<stdlib.h>

#define M (4096)
#define N (8192)

//CUDA kernel for matrix transpose operation
__global__
void m_transpose(float *a, float *a_t) {
	int row = blockIdx.y * blockDim.y  + threadIdx.y;
	int col = blockIdx.x * blockDim.x  + threadIdx.x;

	int index1 = (row * N) + col;
	int index2 = (col * M) + row;

	if (row < M && col < N) {
		a_t[index2] = a[index1];
	}

	return;
}

int main(int argc, char **argv) {
	float *A, *A_T;
	float max_err = 0.0f;
	int size = M * N * sizeof(float);
	dim3 threads_per_blk(16, 16, 1);
	dim3 blocks_per_grid((N+threads_per_blk.x-1)/threads_per_blk.x,
											 (M+threads_per_blk.y-1)/threads_per_blk.y,
												1);

	//Seed the random number generator
	srand(time(NULL));

	//Allocate unified memory for matrices A and A_T
	hipMallocManaged(&A, size);
	hipMallocManaged(&A_T, size);

	//Initialize matrix A (MxN)
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			int index = i * N + j;
			A[index] = float(rand() % 100000 + 1.0f);
		}
	}

	//Launch transpose kernel on GPU
	m_transpose<<<blocks_per_grid, threads_per_blk>>>(A, A_T);

	//Wait for GPU to finish
	hipDeviceSynchronize();

	//Verify GPU transpose operation
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			int index1 = i * N + j;
			int index2 = j * M + i;
			max_err = fabs(fmax(max_err, (A[index1] - A_T[index2])));
		}
	}

	printf("Max error: %f\n", max_err);

	//Free unified memory
	hipFree(A);
	hipFree(A_T);

	return 0;
}
