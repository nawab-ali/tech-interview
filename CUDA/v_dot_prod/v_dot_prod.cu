//Parallel vector dot product on GPU using CUDA


#include <hip/hip_runtime.h>
#include<math.h>
#include<stdio.h>

#define N (2048*2048)
#define BLOCKSIZE 512
#define NUMBLOCKS (N/BLOCKSIZE)

//Parallel vector dot product kernel
__global__
void v_dot_prod(float *a, float *b, float *c) {
	__shared__ float t_prod[BLOCKSIZE];
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	//Parallel vector multiplication
	for (int i = index; i < N; i += stride) {
		t_prod[threadIdx.x] = a[i] * b[i];
	}

	//Wait for all threads in the block to complete
	__syncthreads();

	//Serial summation of vector t_prod at block level
	if (threadIdx.x == 0) {
		float t_sum = 0.0;
		for (int i = 0; i < BLOCKSIZE; i++) {
			t_sum += t_prod[i];
		}
		//Add to global sum atomically
		atomicAdd(c, t_sum);
	}

	return;
}

int main(void) {
	float *x;
	float *y;
	float *sum;
	float t_sum = 0.0f;

	//Allocate unified memory for vectors and sum
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));
	hipMallocManaged(&sum, sizeof(float));

	//Initialize vectors
	*sum = 0.0f;
	for (int i = 0; i < N; i++) {
		x[i] = 2.0f;
		y[i] = 3.0f;
	}

	//Run the vector dot product kernel on the GPU
	v_dot_prod<<<NUMBLOCKS, BLOCKSIZE>>>(x, y, sum);

	//Wait for GPU to complete processing
	hipDeviceSynchronize();

	//Verify the GPU results
	for (int i = 0; i < N; i++) {
		t_sum += x[i] * y[i];
	}

	printf("Error: %f\n", fabs(t_sum - *sum));

	//Free memory
	hipFree(x);
	hipFree(y);
	hipFree(sum);

	return 0;
}
